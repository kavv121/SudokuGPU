
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <stdint.h>
#include <cassert>


#define GPU_CHECKERROR( err ) (gpuCheckError( err, __FILE__, __LINE__ ))
static void gpuCheckError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
enum {
    STAT_NOCHG = 0,
    STAT_UPDATED = 1,
    STAT_NOTOK = 2
};

enum {
    SUFLG_PROPAGATE_SINGLE = (1<<0), //propagated the fact that I solved this cell already
};

template<int SIZE>
struct SudokuProblem {
    uint32_t givens[SIZE][SIZE]; //0 if unknown, digit otherwise
};

/* warning: will break on sizes > 32 */
template<int SIZE>
struct SudokuState {
    uint32_t bitstate[SIZE][SIZE];
    uint8_t  work_flag[SIZE][SIZE]; //flags to cache work done
};

template<int SIZE>
void print_state(const SudokuState<SIZE> &s) {
    for(int r=0;r<SIZE;++r) {
        for(int c=0;c<SIZE;++c) {
            fprintf(stderr, "(%d, %d)", r,c);
            for(int t=0;t<SIZE;++t) {
                if(s.bitstate[r][c] & (1u<<t)) {
                    fprintf(stderr, " %d", t+1);
                }
            }
            fprintf(stderr, "\n");
        }
    }
}

template<int RSIZE>
__global__ void simple_cand_elim(SudokuState<RSIZE*RSIZE> *p, int *rc) {
    __shared__ SudokuState<RSIZE*RSIZE> s;
    __shared__ int block_status, block_ok;
    const int r = threadIdx.x;
    const int c = threadIdx.y;
    if(r == 0 && c == 0){block_status = STAT_NOCHG;block_ok = 1;}
    //copy current values
    const uint32_t myval = p->bitstate[r][c];
    s.bitstate[r][c] = myval; 
    s.work_flag[r][c] = p->work_flag[r][c];
    __syncthreads();
    if(myval == 0) {
        block_ok = 0;
        goto ending;
    }
    if(myval & (myval-1)) {
        goto ending;
    }
    if(s.work_flag[r][c] & SUFLG_PROPAGATE_SINGLE) {goto ending;}
    //here we have a singleton!!, atomically update neighbors
    //row
    for(int oc=0;oc<RSIZE*RSIZE;++oc) {
        if(oc != c) {
            uint32_t old = atomicAnd(&s.bitstate[r][oc], ~myval);
            if((old & (~myval)) != old) {
                block_status = STAT_UPDATED;
            }
        }
    }
    //column
    for(int row=0;row<RSIZE*RSIZE;++row) {
        if(row != r) {
            uint32_t old = atomicAnd(&s.bitstate[row][c], ~myval);
            if((old & (~myval)) != old) {
                block_status = STAT_UPDATED;
            }
        }
    }
    //block
    {
        int baser = RSIZE*(r/RSIZE);
        int basec = RSIZE*(c/RSIZE);
        for(int dr=0;dr<RSIZE;++dr) {
            for(int dc=0;dc<RSIZE;++dc) {
                if(baser + dr == r && basec + dc == c) {
                    continue;
                }
                uint32_t old = atomicAnd(&s.bitstate[baser+dr][basec+dc], ~myval);
                if((old & (~myval)) != old) {
                    block_status = STAT_UPDATED;
                }
            }
        }
    }
    p->work_flag[r][c] |= SUFLG_PROPAGATE_SINGLE;
ending:;
    __syncthreads();
    p->bitstate[r][c] = s.bitstate[r][c];
    if(r == 0 && c == 0) {
        if(!block_ok) {
            block_status = STAT_NOTOK;
        }
        *rc = block_status;
    }
}

template<int RSIZE>
__global__ void singleton_search(SudokuState<RSIZE*RSIZE> *p, int *rc) {
    __shared__ SudokuState<RSIZE*RSIZE> s;
    __shared__ int block_status, block_ok;
    const int r = threadIdx.x;
    const int c = threadIdx.y;
    if(r == 0 && c == 0){block_status = STAT_NOCHG;block_ok = 1;}
    //copy current values
    const uint32_t myval = p->bitstate[r][c];
    s.bitstate[r][c] = myval; 
    __syncthreads();


    
    uint32_t finalval = 0;
    bool ok = true;
    //don't bother if this is already a singleton or nothing
    if(myval == 0) {
        ok = false;
        goto ending;
    }
    if(!(myval & (myval-1))) {
        goto ending;
    }
    //we remove bits that are in other things
    //check row
    {
        uint32_t tval = myval;
        for(int cc=0;cc<RSIZE*RSIZE && tval;++cc) {
            if(cc == c){continue;}
            tval &= ~s.bitstate[r][cc];
        }
        if(tval) {
            if(finalval == 0 || finalval == tval) {
                finalval = tval;
            }
            else {
                ok = false;
                goto ending;
            }
        }
    }
    //column
    {
        uint32_t tval = myval;
        for(int rr=0;rr<RSIZE*RSIZE && tval;++rr) {
            if(rr == r){continue;}
            tval &= ~s.bitstate[rr][c];
        }
        if(tval) {
            if(finalval == 0 || finalval == tval) {
                finalval = tval;
            }
            else {
                ok = false;
                goto ending;
            }
        }
    }
    //region
    {
        uint32_t tval = myval;
        int base_r = RSIZE*(r/RSIZE);
        int base_c = RSIZE*(c/RSIZE);
        for(int dr=0;dr<RSIZE;++dr) {
            for(int dc=0;dc<RSIZE;++dc) {
                const int nr = base_r + dr;
                const int nc = base_c + dc;
                if(nr == r && nc == c){continue;}
                tval &= ~s.bitstate[nr][nc];

            }
        }
        if(tval) {
            if(finalval == 0 || finalval == tval) {
                finalval = tval;
            }
            else {
                ok = false;
                goto ending;
            }
        }
    }
ending:
    //either we broke something, or we concluded that two values need to fit?
    if(!ok || (finalval != 0 && (finalval & (finalval-1)) != 0)) {
        //we can do this since the change is in one direction,
        //and we sync before reading it
        block_ok = 0;
    }
    else if(finalval != 0) { //implies ok && finalval has 1 bit set
        p->bitstate[r][c] = finalval;
        //we can do this since the change is in one direction,
        //and we sync before reading it
        block_status = STAT_UPDATED;
    }
    __syncthreads();
    if(r == 0 && c == 0) {
        if(!block_ok) {
            block_status = STAT_NOTOK;
        }
        *rc = block_status;
    }
}

template<int SIZE>
void fill_state_from_problem(SudokuState<SIZE> *state, const SudokuProblem<SIZE> &problem) {
    memset(state, 0, sizeof(SudokuState<SIZE>));
    for(int r=0;r<SIZE;++r) {
        for(int c=0;c<SIZE;++c) {
            if(problem.givens[r][c] != 0) {
                assert(problem.givens[r][c] >= 1);
                assert(problem.givens[r][c] <= SIZE);
                state->bitstate[r][c] = (1u << (problem.givens[r][c]-1));
            }
            else {
                state->bitstate[r][c] = (1u << SIZE) - 1;
            }
        }
    }
}

template<int RSIZE>
void check_state(const SudokuState<RSIZE*RSIZE> &s) {
    bool ok = true;
    for(int r=0;r<RSIZE*RSIZE;++r) {
        for(int c=0;c<RSIZE*RSIZE;++c) {
            const uint32_t bs = s.bitstate[r][c];
            if(bs == 0 || (bs & (bs-1)) != 0) {
                std::cerr << "Row " << r << " " << "col " << c << " not singleton!" << std::endl;
                ok = false;
            }
        }
    }
    if(!ok){return;}
    //row check
    const uint32_t GOAL = (1u << (RSIZE*RSIZE)) - 1;
    for(int r=0;r<RSIZE*RSIZE;++r) {
        uint32_t xx = 0;
        for(int i=0;i<9;++i) {
            const int nr = r;
            const int nc = i;
            xx |= s.bitstate[nr][nc];
        }
        if(xx != GOAL) {
            std::cerr << "Row " << r << " is no good" << std::endl;
            ok = false;
            break;
        }
    }
    if(!ok){return;}
    for(int c=0;c<RSIZE*RSIZE;++c) {
        uint32_t xx = 0;
        for(int i=0;i<9;++i) {
            const int nr = i;
            const int nc = c;
            xx |= s.bitstate[nr][nc];
        }
        if(xx != GOAL) {
            std::cerr << "Col " << c << " is no good" << std::endl;
            ok = false;
            break;
        }
    }
    if(!ok){return;}
    for(int br=0;br<RSIZE && ok;++br) {
        for(int bc=0;bc<RSIZE;++bc) {
            uint32_t xx = 0;
            for(int i=0;i<9;++i) {
                const int nr = RSIZE*br + (i/3);
                const int nc = RSIZE*bc + (i%3);
                xx |= s.bitstate[nr][nc];
            }
            if(xx != GOAL) {
                std::cerr << "Region " << br << "," << bc << " is no good" << std::endl;
                ok = false;
                break;
            }
        }
    }
    if(!ok){return;}
    std::cerr << "ALL GOOD!" << std::endl;
}

void test_basics(SudokuState<9> &state) {
    SudokuState<9> *d_state;
    int *d_rc;
    int h_rc;
    GPU_CHECKERROR(hipMalloc((void **)&d_state, 
                              sizeof(SudokuState<9>)));
    GPU_CHECKERROR(hipMalloc((void **)&d_rc, 
                              sizeof(int)));
    GPU_CHECKERROR(hipMemset(d_rc, 0, sizeof(int)));

    GPU_CHECKERROR(hipMemcpy(d_state, &state, sizeof(SudokuState<9>), hipMemcpyHostToDevice));
    dim3 num_block(1,1,1);
    dim3 threads_per_block(9,9,1);
    for(h_rc = STAT_UPDATED;h_rc == STAT_UPDATED;)
    {
        h_rc = STAT_NOCHG;
        GPU_CHECKERROR(hipMemset(d_rc, 0, sizeof(int)));
        //singleton_search<3><<<num_block, threads_per_block>>>(d_state, d_rc);
        simple_cand_elim<3><<<num_block, threads_per_block>>>(d_state, d_rc);
        GPU_CHECKERROR(hipGetLastError());
        GPU_CHECKERROR(hipMemcpy(&h_rc, d_rc, sizeof(int), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
        fprintf(stderr, "SIMPLE - GOT RC %d\n", h_rc);
        if(h_rc != STAT_NOCHG){continue;}

        singleton_search<3><<<num_block, threads_per_block>>>(d_state, d_rc);
        GPU_CHECKERROR(hipGetLastError());
        GPU_CHECKERROR(hipMemcpy(&h_rc, d_rc, sizeof(int), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
        fprintf(stderr, "SINGLETON - GOT RC %d\n", h_rc);
        if(h_rc != STAT_NOCHG){continue;}

    }
    GPU_CHECKERROR(hipMemcpy(&state, d_state, sizeof(SudokuState<9>), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    hipFree(d_state);
    hipFree(d_rc);
    

    
    //print_state(state);
    check_state<3>(state);
}


int main(int argc, char **argv) {
    std::string s;
    std::cin >> s;
    if(s.size() != 81) {
        std::cerr << "NEED 81 cells" << std::endl;
    }
    SudokuProblem<9> problem;
    memset(&problem, 0, sizeof(problem));
    for(int i=0;i<s.size();++i)
    {
        if(s[i] >= '1' && s[i] <= '9') {
            int dig = s[i] - '0';
            int r = i/9;
            int c = i % 9;
            problem.givens[r][c] = dig;
        }
    }
    SudokuState<9> mystate;
    fill_state_from_problem(&mystate, problem);
    print_state(mystate);
    test_basics(mystate);
    return 0;
}
